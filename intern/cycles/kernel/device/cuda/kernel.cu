
#include <hip/hip_runtime.h>
/* SPDX-License-Identifier: Apache-2.0
 * Copyright 2011-2022 Blender Foundation */

/* CUDA kernel entry points */

#ifdef __CUDA_ARCH__

#  include "kernel/device/cuda/compat.h"
#  include "kernel/device/cuda/config.h"
#  include "kernel/device/cuda/globals.h"

#  include "kernel/device/gpu/image.h"
#  include "kernel/device/gpu/kernel.h"

#endif
