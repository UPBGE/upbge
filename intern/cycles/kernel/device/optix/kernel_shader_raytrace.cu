#include "hip/hip_runtime.h"
/* SPDX-License-Identifier: Apache-2.0
 * Copyright 2021-2022 Blender Foundation. */

/* Copy of the regular kernels with additional shader ray-tracing kernel that takes
 * much longer to compiler. This is only loaded when needed by the scene. */

#include "kernel/device/optix/kernel.cu"

#include "kernel/integrator/shade_surface.h"

extern "C" __global__ void __raygen__kernel_optix_integrator_shade_surface_raytrace()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (kernel_params.path_index_array) ? kernel_params.path_index_array[global_index] :
                                                       global_index;
  integrator_shade_surface_raytrace(nullptr, path_index, kernel_params.render_buffer);
}

extern "C" __global__ void __raygen__kernel_optix_integrator_shade_surface_mnee()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (kernel_params.path_index_array) ? kernel_params.path_index_array[global_index] :
                                                       global_index;
  integrator_shade_surface_mnee(nullptr, path_index, kernel_params.render_buffer);
}
