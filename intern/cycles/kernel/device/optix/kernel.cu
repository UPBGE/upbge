#include "hip/hip_runtime.h"
/* SPDX-License-Identifier: Apache-2.0
 * Copyright 2019, NVIDIA Corporation.
 * Copyright 2019-2022 Blender Foundation. */

// clang-format off
#include "kernel/device/optix/compat.h"
#include "kernel/device/optix/globals.h"

#include "kernel/device/gpu/image.h"  /* Texture lookup uses normal CUDA intrinsics. */

#include "kernel/tables.h"

#include "kernel/integrator/state.h"
#include "kernel/integrator/state_flow.h"
#include "kernel/integrator/state_util.h"

#include "kernel/integrator/intersect_closest.h"
#include "kernel/integrator/intersect_shadow.h"
#include "kernel/integrator/intersect_subsurface.h"
#include "kernel/integrator/intersect_volume_stack.h"
// clang-format on

extern "C" __global__ void __raygen__kernel_optix_integrator_intersect_closest()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (kernel_params.path_index_array) ?
                             kernel_params.path_index_array[global_index] :
                             global_index;
  integrator_intersect_closest(nullptr, path_index, kernel_params.render_buffer);
}

extern "C" __global__ void __raygen__kernel_optix_integrator_intersect_shadow()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (kernel_params.path_index_array) ?
                             kernel_params.path_index_array[global_index] :
                             global_index;
  integrator_intersect_shadow(nullptr, path_index);
}

extern "C" __global__ void __raygen__kernel_optix_integrator_intersect_subsurface()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (kernel_params.path_index_array) ?
                             kernel_params.path_index_array[global_index] :
                             global_index;
  integrator_intersect_subsurface(nullptr, path_index);
}

extern "C" __global__ void __raygen__kernel_optix_integrator_intersect_volume_stack()
{
  const int global_index = optixGetLaunchIndex().x;
  const int path_index = (kernel_params.path_index_array) ?
                             kernel_params.path_index_array[global_index] :
                             global_index;
  integrator_intersect_volume_stack(nullptr, path_index);
}

